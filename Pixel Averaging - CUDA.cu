#include "hip/hip_runtime.h"
//Submission should be named as  <RollNumber>_Prog.cu
//Upload just this cu file and nothing else. If you upload it as a zip, it will not be evaluated. 


#include <stdio.h>
#define M 514 
//Input has 514 rows and columns 

#define N 512 
//For output, only 512 rows and columns need to be computed. 


//TODO: WRITE GPU KERNEL. It should not be called repeatedly from the host, but just once. Each time it is called, it may process more than pixel or not //process any pixel at all. 

//Code to proces 514*514 input elements using 9*48 threads
//Additional checks included to skip processing boundary elements (only 512*512 to be computed)
// i> 513 : Skip Top row ; i < 263682 : Skip Bottom row 
// i%514=0 : Skip left most column ; (i+1)%514=0 : Skip Right most column

__global__ void computeOutput( int *a, int *b, int size) {
	int numThrds = 9*48;
	int threadID = threadIdx.x + blockIdx.x * blockDim.x;
	for (int i=threadID;i<size;i+=numThrds) {
		if ((i > 513) && (i < 263682 ) && (i%514 != 0 ) & ((i+1)%514 != 0) ){
			b[i] = (a[i-1]+a[i+1]+a[i-514]+a[i+514])/4;
		}
	}
}

main (int argc, char **argv) {
  int A[M][M], B[M][M];
  int *d_A, *d_B; // These are the copies of A and B on the GPU
  int *h_B;       // This is a host copy of the output of B from the GPU
  int i, j;

  // Input is randomly generated
  for(i=0;i<M;i++) {
    for(j=0;j<M;j++) {
      A[i][j] = rand()/1795831;
      //printf("%d\n",A[i][j]);
    }
  }

  // sequential implementation of main computation
  for(i=1;i<M-1;i++) {
    for(j=1;j<M-1;j++) {
      B[i][j] = (A[i-1][j]+A[i+1][j]+A[i][j-1]+A[i][j+1])/4;
    }
  }


  // TODO: ALLOCATE MEMORY FOR GPU COPIES OF d_A AND d_B
  hipMalloc((void **) &d_A, M*M*sizeof(int));
  hipMalloc((void **) &d_B, M*M*sizeof(int));
  
  int insize = M*M*sizeof(int);

  // TODO: COPY A TO d_A
  hipMemcpy(d_A, A, insize, hipMemcpyHostToDevice);

  // TODO: CREATE BLOCKS with THREADS AND INVOKE GPU KERNEL
  //Use 9 blocks, each with 48 threads
  computeOutput<<<9,48>>>(d_A,d_B,M*M);
  hipDeviceSynchronize();

  // TODO: COPY d_B BACK FROM GPU to CPU in variable h_B
  h_B = (int *) malloc(insize);
  hipMemcpy(h_B, d_B, insize, hipMemcpyDeviceToHost);

  // TODO: Verify result is correct by comparing
  int error_cnt=0;
  for(i=1;i<M-1;i++) {
    for(j=1;j<M-1;j++) {
		//print only those elements for which the above subtraction is non-zero
		if (B[i][j] - h_B[i*514+j] != 0) {
			printf("i: %d ,j: %d , B[i][j]: %d , h_B[i*514+j]: %d \n", i,j,B[i][j],h_B[i*514+j]);
			error_cnt=error_cnt+1;
		}
    }
  }
  

  //IF even one element of h_B and B differ, report an error.
  //Otherwise, there is no error.
  //If your program is correct, no error should occur.
  if (error_cnt>1) {
	printf("Error: Found %d discrepancies between CPU and GPU calculation \n", error_cnt);
  } 
  else {
	printf("Code completed successfully! \n");
  }	
}
   

/*Remember the following guidelines to avoid losing marks
Index of an array should not exceed the array size. 
Do not ignore the fact that boundary rows and columns need not be computed (in fact, they cannot be computed since they don't have four neighbors)
No output array-element should be computed more than once
No marks will be given if the program does not compile or run (TAs will not debug your program at all)
*/

